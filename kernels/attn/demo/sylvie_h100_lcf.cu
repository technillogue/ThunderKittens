#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;
template<int D, int NUM_WORKERS> struct attn_fwd_layout {
    using qo_tile   = st_bf<64, D>;
    using kv_tile   = st_bf<D==64?192:128, D>;
    using qo_global = kittens::gl<bf16, -1, -1, -1, D, qo_tile>;
    using kv_global = kittens::gl<bf16, -1, -1, -1, D, kv_tile>;
    struct globals { qo_global O, Q; kv_global K, V; };
    struct input_block    { kv_tile k, v; };
    struct scratch_block  { qo_tile q[NUM_WORKERS]; };
    struct common_state   { int batch, head, seq, q_start_idx; };
    struct consumer_state {
        rt_fl<16, qo_tile::cols> o_reg;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec, norm_vec;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec_last_scaled, max_vec_scaled;
        rt_fl<16, kv_tile::rows> att_block;
        rt_bf<16, kv_tile::rows> att_block_mma;
    };
};
template<int D, int WINDOW_SIZE = 256> struct attn_fwd_template {
    static constexpr int NUM_CONSUMER_WARPS = 12, NUM_WORKERS = NUM_CONSUMER_WARPS/4, INPUT_PIPE_STAGES = 2;
    using layout = attn_fwd_layout<D, NUM_WORKERS>;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int task_id = gridDim.x*args.task_iter + blockIdx.x;
        int seq_q = (args.globals.Q.rows + NUM_WORKERS*layout::qo_tile::rows - 1)/(NUM_WORKERS*layout::qo_tile::rows);
        args.common.batch = task_id / (seq_q*args.globals.K.depth); task_id -= args.common.batch * seq_q * args.globals.K.depth;
        args.common.head  = task_id / seq_q;                        task_id -= args.common.head  * seq_q;
        args.common.seq   = task_id;
        args.common.q_start_idx = task_id * NUM_WORKERS * layout::qo_tile::rows;
        args.num_iters = args.common.batch < args.globals.Q.batch ? (args.globals.K.rows + layout::kv_tile::rows - 1)/(layout::kv_tile::rows) : -1;
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {
            warpgroup::producer_registers();
        }
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                tma::load_async(args.input.k, args.globals.K, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
                tma::load_async(args.input.v, args.globals.V, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
            }
            else if(laneid() == 0) arrive(args.inputs_arrived);
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            warpgroup::consumer_registers<NUM_WORKERS>();
            if((args.common.seq*NUM_WORKERS + warpgroup::groupid())*layout::qo_tile::rows < args.globals.Q.rows) // out of bounds?
                warpgroup::load(args.scratch.q[warpgroup::groupid()], args.globals.Q,
                                {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
            zero(args.state.o_reg);
            zero(args.state.norm_vec);
            neg_infty(args.state.max_vec);
            warpgroup::sync(warpgroup::groupid());
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            constexpr float TEMPERATURE_SCALE = (D == 128) ? 0.08838834764f*1.44269504089f : 0.125f*1.44269504089f;

            // Calculate current query position
            int q_idx_base = args.common.q_start_idx + warpgroup::groupid() * layout::qo_tile::rows;
            // Calculate current key positions for this tile
            int k_idx_start = args.iter * layout::kv_tile::rows;
            int k_idx_end = min(k_idx_start + layout::kv_tile::rows, (int)args.globals.K.rows);


            // A = Q @ K.T
            warpgroup::mm_ABt(args.state.att_block, args.scratch.q[warpgroup::groupid()], args.input.k);
            mul(args.state.max_vec_last_scaled, args.state.max_vec, TEMPERATURE_SCALE);
            warpgroup::mma_async_wait();
            // Apply sliding window mask, each row is a query position
            // not sure why 16
            #pragma unroll
            for (int q_row = 0; q_row < 16; q_row++) {
                 int q_pos = q_idx_base + q_row;
                 // window boundaries
                 int window_start = max(0, q_pos - WINDOW_SIZE/2);
                 int window_end = min((int)args.globals.K.rows, q_pos + WINDOW_SIZE/2 + 1);
                 // for each key in the current tile
                 #pragma unroll
                 for (int k_col = 0; k_col < layout::kv_tile::rows; k_col++) {
                     int k_pos = k_idx_start + k_col;
                     // if the key is beyond valid keys or outside window
                     if (k_pos >= k_idx_end || k_pos < window_start || k_pos >= window_end) {
                        float neginf = base_types::constants<float>::neg_infty();
                        args.state.att_block.tiles[q_row/4][k_col/4].data[q_row%4 * layout::kv_tile::cols + k_col%4] = float2(neginf, neginf);
                     }
                 }
            }


            // softmax
            right_fill(args.state.att_block, args.state.att_block, args.globals.K.rows - args.iter*layout::kv_tile::rows, base_types::constants<float>::neg_infty());
            row_max(args.state.max_vec, args.state.att_block, args.state.max_vec); // accumulate onto the max_vec
            mul(args.state.max_vec_scaled, args.state.max_vec, TEMPERATURE_SCALE);
            mul(args.state.att_block, args.state.att_block, TEMPERATURE_SCALE);
            sub_row(args.state.att_block, args.state.att_block, args.state.max_vec_scaled);
            exp2(args.state.att_block, args.state.att_block);
            sub(args.state.max_vec_last_scaled, args.state.max_vec_last_scaled, args.state.max_vec_scaled);
            exp2(args.state.max_vec_last_scaled, args.state.max_vec_last_scaled);
            mul(args.state.norm_vec, args.state.norm_vec, args.state.max_vec_last_scaled);
            row_sum(args.state.norm_vec, args.state.att_block, args.state.norm_vec); // accumulate onto the norm_vec
            mul_row(args.state.o_reg, args.state.o_reg, args.state.max_vec_last_scaled); // normalize o_reg before mma
            copy(args.state.att_block_mma, args.state.att_block); // convert to bf16 for mma
            // O += A @ V
            warpgroup::mma_AB(args.state.o_reg, args.state.att_block_mma, args.input.v);
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished); // done!
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            if((args.common.seq*NUM_WORKERS+warpgroup::groupid())*64 < args.globals.Q.rows) { // out of bounds?
                div_row(args.state.o_reg, args.state.o_reg, args.state.norm_vec);
                auto &o_smem = reinterpret_cast<typename layout::qo_tile&>(args.scratch.q[warpgroup::groupid()]);
                warpgroup::store(o_smem, args.state.o_reg);
                warpgroup::sync(warpgroup::groupid());
                if(warpgroup::warpid() == 0)
                    tma::store_async(args.globals.O, o_smem, {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
                tma::store_async_read_wait();
            }
            __syncwarp();
            if(laneid() == 0) arrive(args.finish_finished); // done!
        }
    };
};
// kernel is kittens::prototype::lcf::kernel<attn_fwd_template<HEAD_DIM>>;

PYBIND11_MODULE(window_attn, m) {
    m.doc() = "window attention :)";
    py::bind_kernel<lcf::kernel<attn_fwd_template<128>>>(m, "attn_fwd",
        &layout::globals::O, &layout::globals::Q, &layout::globals::K, &layout::globals::V
    );
}