#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include "static_switch.h"
#include <iostream>

constexpr int NUM_CONSUMERS = (2); 
constexpr int NUM_PRODUCERS = (1);

constexpr int NCTA = 2;
constexpr int CLUSTER_DIM = 2;
constexpr int PIPE_DEPTH = 4;

using namespace kittens;

template<int Mb, int Nb, int Kb>
struct matmul_config_t {
    static constexpr int Mb_ = Mb;
    static constexpr int Nb_ = Nb;
    static constexpr int Kb_ = Kb;
};

template <typename Config>
struct matmul_globals {
    using a_tile = st_fl8_e4m3<Config::Mb_, Config::Kb_>;
    using b_tile = st_fl8_e4m3<Config::Nb_/CLUSTER_DIM, Config::Kb_>;
    using d_tile = st_hf<Config::Mb_, Config::Nb_/PIPE_DEPTH>;

    using a_gl = gl<fp8e4m3, 1, 1, -1, -1, a_tile>;
    using b_gl = gl<fp8e4m3, 1, 1, -1, -1, b_tile>;
    using d_gl = gl<half,    1, 1, -1, -1, d_tile>;

    a_gl a;
    b_gl b;
    d_gl d;
};

constexpr int NUM_WORKERS = (NUM_CONSUMERS + NUM_PRODUCERS) * 4;
constexpr int NUM_THREADS = NUM_WORKERS * kittens::WARP_THREADS;

template <typename Config>
__device__ static inline int get_iters_per_task(const matmul_globals<Config> &g) {
    return g.a.cols / Config::Kb_;
}

template<typename Config, int SUPER_M=8>
__device__ static inline int2 get_task_idx(const matmul_globals<Config> &g, int task_iter, bool is_consumer) {
    constexpr int CLUSTER_M = 4*Config::Mb_, CLUSTER_N = Config::Nb_;
    int cluster_x = clusterIdx().x, ctarank = cluster_ctarank();
    int task_id = task_iter * (gridDim.x/2) + cluster_x;
    int Rblocks = g.d.rows / CLUSTER_M, Cblocks = g.d.cols / CLUSTER_N;
    int super_rows = (Rblocks/SUPER_M)*SUPER_M,
        final_rows = Rblocks - super_rows,
        super_repeat = SUPER_M*Cblocks;
    if (task_id < super_rows * Cblocks) {
        return { 
            (SUPER_M*(task_id/super_repeat) + task_id%SUPER_M)*4 + ctarank*2 + is_consumer*(warpgroup::groupid()),
            is_consumer ? (task_id%super_repeat)/SUPER_M : 2*((task_id%super_repeat)/SUPER_M) + ctarank
        };
    }
    else if (task_id < Rblocks*Cblocks) {
        int remainder_id = task_id - super_rows*Cblocks;
        return {
            (super_rows + remainder_id%final_rows)*4 + ctarank*2 + is_consumer*(warpgroup::groupid()),
            is_consumer ? remainder_id/final_rows : 2*(remainder_id/final_rows) + ctarank
        };
    }
    else {
        return { -1, -1 };
    }
}

template <typename Config>
__global__ __cluster_dims__(CLUSTER_DIM) __launch_bounds__(NUM_THREADS, 1)
void matmul(const __grid_constant__ matmul_globals<Config> g) {

    extern __shared__ int __shm[]; 
    tma_swizzle_allocator al((int*)&__shm[0]);
    int warpid = kittens::warpid(), warpgroupid = warpgroup::groupid();
    int iters_per_task = get_iters_per_task(g);

    using a_tile = matmul_globals<Config>::a_tile;
    using b_tile = matmul_globals<Config>::b_tile;
    using d_tile = matmul_globals<Config>::d_tile;
    
    a_tile (&a_smem)[PIPE_DEPTH][NUM_CONSUMERS] = al.allocate<a_tile, PIPE_DEPTH, NUM_CONSUMERS>();
    b_tile (&b_smem)[PIPE_DEPTH]                = al.allocate<b_tile, PIPE_DEPTH>();
    d_tile (&d_smem)                            = al.allocate<d_tile>();

    tma::cluster::sync();
    auto all_tmem = allocate_tmem<1, NCTA>();
    using d_tmem_t = tmem<float, Config::Mb_, Config::Nb_>;

    __shared__ kittens::semaphore inputs_arrived[PIPE_DEPTH], inputs_finished[PIPE_DEPTH], outputs_arrived, outputs_finished[NUM_CONSUMERS];
    uint32_t bitfield = 0xFFFF0000; // ***_finished phase bits start as 1s, ***_arrived phase bits start as 0s

    if (threadIdx.x == 0) { 
        for(int i = 0; i < PIPE_DEPTH; i++) {
            init_semaphore(inputs_arrived[i], 0, 2); 
            init_semaphore(inputs_finished[i], 0, NUM_CONSUMERS); 
        }
        init_semaphore(outputs_arrived, 0, 1);
        for(int i = 0; i < NUM_CONSUMERS; i++) {
            init_semaphore(outputs_finished[i], 0, 1);
        }
    }

    tma::cluster::sync();
    
    if(warpgroupid == NUM_CONSUMERS) {
        warpgroup::decrease_registers<56>();
        int ctarank = cluster_ctarank(); 
        if(warpgroup::warpid() == 3) {
            int input_ring = 0; // tracking which input block is being loaded
            for(int task_iter = 0; true; task_iter++) {
                int2 rowcol = get_task_idx<Config>(g, task_iter, false);
                if(rowcol.x == -1) {
                    for(int idx = 0; idx < (PIPE_DEPTH); idx++) {
                        tma::cluster::wait(inputs_finished[input_ring], prototype::get_phasebit<1>(bitfield, input_ring));
                        input_ring=prototype::ring_advance<PIPE_DEPTH>(input_ring);
                    }
                    if(laneid() == 0) arrive(outputs_arrived);
                    return;
                }
                for (int idx = 0; idx < iters_per_task; idx++) {
                    tma::cluster::wait(inputs_finished[input_ring], prototype::get_phasebit<1>(bitfield, input_ring));
                    prototype::update_phasebit<1>(bitfield, input_ring);
                    if(task_iter>0 && idx==PIPE_DEPTH-1 && laneid() == 0) arrive(outputs_arrived); 
                    tma::cluster::expect(inputs_arrived[input_ring], 0, a_smem[0][0], a_smem[0][1], b_smem[0]);
                    tma::cluster::load_async(a_smem[input_ring][0], g.a, {(rowcol.x+0), idx}, inputs_arrived[input_ring], (uint16_t)(1<<ctarank), 0);
                    tma::cluster::load_async(a_smem[input_ring][1], g.a, {(rowcol.x+1), idx}, inputs_arrived[input_ring], (uint16_t)(1<<ctarank), 0);
                    tma::cluster::load_async(b_smem[input_ring],    g.b, { rowcol.y,    idx}, inputs_arrived[input_ring], (uint16_t)(1<<ctarank), 0);
                    input_ring=prototype::ring_advance<PIPE_DEPTH>(input_ring);
                }
            }
        }
        else if(ctarank == 0 && (warpgroup::warpid() == 0 || warpgroup::warpid() == 1)) { // launch the MMA's
            d_tmem_t d_tmem = all_tmem.subtile<d_tmem_t>(0, warpgroup::warpid()*Config::Nb_);
            int input_ring = 0; // tracking which input block is being loaded
            for(int task_iter = 0; true; task_iter++) {
                int2 rowcol = get_task_idx<Config>(g, task_iter, false);
                if(rowcol.x == -1) return;
                tma::cluster::wait(outputs_finished[warpgroup::warpid()], (task_iter+1)%2); // make sure tensor memory is ready to be written to.
                tma::cluster::wait(inputs_arrived[input_ring], prototype::get_phasebit<0>(bitfield, input_ring));
                prototype::update_phasebit<0>(bitfield, input_ring);
                mm2_ABt(d_tmem, a_smem[0][warpgroup::warpid()], b_smem[0], inputs_finished[0]);
                input_ring=prototype::ring_advance<PIPE_DEPTH>(input_ring);
                for(int idx = 1; idx < iters_per_task; idx++) {
                    tma::cluster::wait(inputs_arrived[input_ring], prototype::get_phasebit<0>(bitfield, input_ring));
                    prototype::update_phasebit<0>(bitfield, input_ring);
                    mma2_ABt(d_tmem, a_smem[input_ring][warpgroup::warpid()], b_smem[input_ring], inputs_finished[input_ring]);
                    input_ring=prototype::ring_advance<PIPE_DEPTH>(input_ring);
                }
            }
        }
    }
    else {
        warpgroup::increase_registers<224>();
        d_tmem_t d_tmem = all_tmem.subtile<d_tmem_t>(0, warpgroupid*Config::Nb_);
        for(int task_iter = 0; true; task_iter++) {
            int2 rowcol = get_task_idx<Config>(g, task_iter, true);
            if(rowcol.x == -1) return;
            kittens::wait(outputs_arrived, task_iter%2);
            rt_hf<Config::Mb_/4, d_tile::cols> d_reg[4];
            if(warpgroupid == 1) group<8>::sync(15);
            #pragma unroll
            for(int i = 0; i < Config::Nb_/d_tile::cols; i++) {
                warpgroup::load_async(d_reg[i], d_tmem.template subtile<tmem<float, Config::Mb_, Config::Nb_/PIPE_DEPTH>>(0, Config::Nb_/PIPE_DEPTH*i));
            }
            tm_load_wait();
            warpgroup::sync(warpgroupid);
            if(warpgroup::laneid() == 0) arrive(outputs_finished[warpgroupid]); // Tensor memory for warpgroup 0 is now free.
            if(warpgroupid == 0) group<8>::sync(15);
            if(warpgroupid == 1) group<8>::sync(14);
            warpgroup::store(d_smem, d_reg[0]);
            warpgroup::sync(warpgroupid);
            if(warpgroup::warpid() == 0) tma::store_async(g.d, d_smem, {rowcol.x, 4*rowcol.y+0});
            #pragma unroll
            for(int i = 1; i < Config::Nb_/d_tile::cols; i++) {
                tma::store_async_read_wait();
                warpgroup::sync(warpgroupid);
                warpgroup::store(d_smem, d_reg[i]);
                warpgroup::sync(warpgroupid);
                if(warpgroup::warpid() == 0) tma::store_async(g.d, d_smem, {rowcol.x, 4*rowcol.y+i});
            }
            tma::store_async_read_wait();
            if(warpgroupid == 0) group<8>::sync(14);
            group<8>::sync(15); // All consumers sync here.
        }
    }
}


constexpr bool NCU = false;
#include <iostream>
#include <random>
#include <hip/hip_bf16.h>
#include <omp.h>

void cpu_gemm(float* a, float* b, float* c, int M, int N, int K) {
    #pragma omp parallel for collapse(2) // otherwise the CPU version takes for everrrrrr
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += a[i * K + k] * b[j * N + k];
            }
            c[i * N + j] = sum;
        }
    }
}

template<int M, int N, int K>
struct runtime_config {
    static constexpr int Mb_ = M;
    static constexpr int Nb_ = N;
    static constexpr int Kb_ = K;
};

template <typename Config>
void dispatch_matmul(fp8e4m3 *d_A, fp8e4m3 *d_B, half *d_C, size_t M, size_t N, size_t K, dim3 grid, dim3 block) {
    using globals = matmul_globals<Config>;
    typename globals::a_gl Ag{d_A, nullptr, nullptr, M, K};
    typename globals::b_gl Bg{d_B, nullptr, nullptr, N, K};
    typename globals::d_gl Dg{d_C, nullptr, nullptr, M, N};
    globals G{Ag, Bg, Dg};
    matmul<<<grid, block, MAX_SHARED_MEMORY-1024>>>(G);
}

template <typename Config>
int run_benchmark(size_t M, size_t N, size_t K) {
    hipError_t cudaStatus;

    std::cout << "--------------------  M=" << M << " N=" << N << " K=" << K << "  --------------------\n";
    std::cout << "Block size: " << Config::Mb_ << "x" << Config::Nb_<< "\n";

    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    float *h_C_ref = new float[M * N];

    std::cout << "Allocated host memory" << std::endl;

    // Initialize random number generator
    std::random_device rd;
    std::mt19937 gen(42);
    std::uniform_real_distribution<> dis(-0.5, 0.5);

    // Initialize matrices with random values
    for (int i = 0; i < M * K; ++i) h_A[i] = dis(gen);
    for (int i = 0; i < K * N; ++i) h_B[i] = dis(gen);

    std::cout << "Initialized matrices" << std::endl;

    // Allocate device memory
    fp8e4m3 *d_A, *d_B;
    half *d_C;
    hipMalloc(&d_A, M*K*sizeof(fp8e4m3));
    hipMalloc(&d_B, K*N*sizeof(fp8e4m3));
    hipMalloc(&d_C, M*N*sizeof(half));

    // Check for CUDA errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Optionally, you might want to exit the program or handle the error in some way
        return -1;
    }

    std::cout << "Allocated device memory" << std::endl;

    // Convert to __hip_bfloat16 and copy to device
    fp8e4m3 *h_A_fp8 = new fp8e4m3[M * K];
    fp8e4m3 *h_B_fp8 = new fp8e4m3[K * N];
    for (int i = 0; i < M * K; ++i) h_A_fp8[i] = fp8e4m3(h_A[i]);
    for (int i = 0; i < K * N; ++i) h_B_fp8[i] = fp8e4m3(h_B[i]);
    for (int i = 0; i < M * K; ++i) h_A[i] = float(h_A_fp8[i]);
    for (int i = 0; i < K * N; ++i) h_B[i] = float(h_B_fp8[i]);

    hipMemcpy(d_A, h_A_fp8, M*K*sizeof(fp8e4m3), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_fp8, K*N*sizeof(fp8e4m3), hipMemcpyHostToDevice);

    std::cout << "Copied matrices to device" << std::endl;

    // Perform CPU matrix multiplication for reference
    if(true) cpu_gemm(h_A, h_B, h_C_ref, M, N, K);

    std::cout << "Performed CPU matrix multiplication" << std::endl;

    unsigned long mem_size = MAX_SHARED_MEMORY - 1024;
    hipFuncSetAttribute(reinterpret_cast<const void*>(matmul<Config>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

    // Launch kernel
    dim3 grid(148, 1);
    dim3 block(NUM_THREADS);
    std::cout << "Launching warmup kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
    for(int i = 0; i < (NCU ? 1 : 1); i++) { // warmup
        dispatch_matmul<Config>(d_A, d_B, d_C, M, N, K, grid, block);
    }

    // Start timing
    hipDeviceSynchronize();
    std::cout << "Launching kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
    auto start = std::chrono::high_resolution_clock::now();

    constexpr int ITERS = (NCU ? 1 : 5);
    for(int i = 0; i < ITERS; i++) {
        dispatch_matmul<Config>(d_A, d_B, d_C, M, N, K, grid, block);
    }
    hipDeviceSynchronize();

    // End timing
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    std::chrono::duration<double> diff = end - start;
    double useconds = diff.count() * 1e6 / ITERS;

    // Calculate TFLOPs
    double flops = double(2.0) * M * N * K; // 2 FLOPs per multiply-add
    double tflops = (flops / useconds) / 1e6;

    std::cout << "Avg Kernel execution time: " << useconds << " us\n";
    std::cout << "Achieved performance: " << tflops << " TFLOPs\n";
    
    // Check for CUDA errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Optionally, you might want to exit the program or handle the error in some way
        return -1;
    }

    // Copy result back to host
    half *h_C_fp16 = new half[M * N];
    hipMemcpy(h_C_fp16, d_C, M*N*sizeof(half), hipMemcpyDeviceToHost);

    std::cout << "Copied result back to host" << std::endl;

    // Convert result back to float for comparison
    for (int i = 0; i < M * N; ++i) h_C[i] = __half2float(h_C_fp16[i]);

    std::cout << "Converted result back to float" << std::endl;

    // Check result
    float max_error = 0.0f;
    int error_count = 0;
    for (int i = 0; i < M * N; ++i) {
        float error = std::abs(h_C[i] - h_C_ref[i]);
        if(error > 1.0) { // large because of bf16 vs fp32 numerics
            if(error_count < 20) std::cout << "Error at row " << i / N << " col " << i % N << ": " << h_C[i] << " != " << h_C_ref[i] << " (ref)" << std::endl;
            else if(error_count == 21) std::cout << "Too many errors to show them all.\n";
            error_count++;
        }
        max_error = std::max(max_error, error);
    }
    
    std::cout << "Max error: " << max_error << std::endl;
    std::cout << "Error count: " << error_count << std::endl;

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_ref;
    delete[] h_A_fp8;
    delete[] h_B_fp8;
    delete[] h_C_fp16;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

int run_matmul_with_config(size_t M, size_t N, size_t K, int tile_m, int tile_n, int tile_k) {
    int result = -1;
    // TILEM_SWITCH(tile_m, [&]() {
        TILEN_SWITCH(tile_n, [&]() {
            TILEK_SWITCH(tile_k, [&]() {
                result = run_benchmark<runtime_config<128, kTileN, kTileK>>(M, N, K);
            });
        });
    // });
    return result;
}

int main() {
    std::vector<std::tuple<int,int,int>> shapes = {
        // {8192, 8192, 8192},
        {16384, 16384, 16384},
        // {128, 14336, 8192},
        // {256, 14336, 8192},
        // {512, 14336, 8192},
        // {1024, 14336, 8192},
        // {4096, 14336, 8192},
        {8192, 14336, 8192}
    };


    std::vector<std::tuple<int, int, int>> configs = {  
        // {64, 64, 64},
        // {64, 64, 128},
        // {64, 128, 64},
        // {64, 128, 128},
        // {64, 256, 64},
        // {64, 256, 128},
        {128, 64, 64},
        {128, 64, 128},
        {128, 128, 64},
        {128, 128, 128},
        {128, 256, 64},
        {128, 256, 128},
    };

    for (const auto& [tile_m, tile_n, tile_k] : configs) {
        std::cout << "\nTesting configuration: TileM=" << tile_m << " TileN=" << tile_n << " TileK=" << tile_k << std::endl;
        for(const auto& [M, N, K] : shapes) {
            std::cout << "Running with shape: M=" << M << " N=" << N << " K=" << K << std::endl;
            run_matmul_with_config(M, N, K, tile_m, tile_n, tile_k);
        }
    }

    return 0;
}
